
#include <hip/hip_runtime.h>
#include <png.h>
#include <zlib.h>

#include <cassert>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <chrono>

using namespace std::chrono;
using namespace std;

#define MASK_N 2
#define MASK_X 5
#define MASK_Y 5
#define SCALE 8



#define CC_GRID_MAX_X_DIM (1 << 31) - 1
#define CC_GRID_MAX_Y_DIM 65535
#define CC_GRID_MAX_Z_DIM 65535

#define CC_BLOCK_MAX_X_DIM 1024
#define CC_BLOCK_MAX_Y_DIM 1024
#define CC_BLOCK_MAX_Z_DIM 64

#define CC_BLOCK_MAX_N_THREADS 1024

#define CC_MAX_N_RSD_BLOCKS 32
#define CC_MAX_N_RSD_WARPS 64
#define CC_MAX_N_RSD_THREADS 2048


// 1.png: 4928 x 3264 x 3
// 2.png: 16320 x 10809 x 3
// 3.png: 634 x 634 x 3

// 4.png: 900 x 622 x 3

// 5.png: 1800 x 1244 x 3
// 6.png: 3600 x 2488 x 3
// 7.png: 7200 x 4976 x 3
// 8.png: 14400 x 9952 x 3


// #define GRID_N_X
// #define GRID_N_Y
#define GRID_N_Z 1

#define BLOCK_N_X 16
#define BLOCK_N_Y 8
#define BLOCK_N_Z 3

// #define BLOCK_N_THREADS




int read_png(const char* filename, unsigned char** image, unsigned* height, unsigned* width,
    unsigned* channels) {
    unsigned char sig[8];
    FILE* infile;
    infile = fopen(filename, "rb");

    fread(sig, 1, 8, infile);
    if (!png_check_sig(sig, 8)) return 1; /* bad signature */

    png_structp png_ptr;
    png_infop info_ptr;

    png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png_ptr) return 4; /* out of memory */

    info_ptr = png_create_info_struct(png_ptr);
    if (!info_ptr) {
        png_destroy_read_struct(&png_ptr, NULL, NULL);
        return 4; /* out of memory */
    }

    png_init_io(png_ptr, infile);
    png_set_sig_bytes(png_ptr, 8);
    png_read_info(png_ptr, info_ptr);
    int bit_depth, color_type;
    png_get_IHDR(png_ptr, info_ptr, width, height, &bit_depth, &color_type, NULL, NULL, NULL);

    png_uint_32 i, rowbytes;
    png_bytep row_pointers[*height];
    png_read_update_info(png_ptr, info_ptr);
    rowbytes = png_get_rowbytes(png_ptr, info_ptr);
    *channels = (int)png_get_channels(png_ptr, info_ptr);

    if ((*image = (unsigned char*)calloc((rowbytes + 4*3) * (*height + 4), 1)) == NULL) {
        png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
        return 3;
    }
 
    for (i = 0; i < *height; ++i) {
        row_pointers[i] = *image + (i + 2) * (rowbytes + 4*3) + 2*3;
    }

    png_read_image(png_ptr, row_pointers);
    png_read_end(png_ptr, NULL);
    png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
    return 0;
}


void write_png(const char* filename, png_bytep image, const unsigned height, const unsigned width,
    const unsigned channels) {
    FILE* fp = fopen(filename, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_set_IHDR(png_ptr, info_ptr, width, height, 8, PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
        PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);
    png_set_filter(png_ptr, 0, PNG_NO_FILTERS);
    png_write_info(png_ptr, info_ptr);
    png_set_compression_level(png_ptr, 0);

    png_bytep row_ptr[height];
    for (int i = 0; i < height; ++i) {
        row_ptr[i] = image + i * width * channels * sizeof(unsigned char);
    }
    png_write_image(png_ptr, row_ptr);
    png_write_end(png_ptr, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    fclose(fp);
}






__global__ void sobel(unsigned char *s, unsigned char *t, 
                                unsigned height, unsigned width, unsigned channels)
{


    int mask[MASK_N][MASK_X][MASK_Y] = {
    
        {{ -1, -4, -6, -4, -1},
        { -2, -8,-12, -8, -2},
        {  0,  0,  0,  0,  0},
        {  2,  8, 12,  8,  2},
        {  1,  4,  6,  4,  1}},

        {{ -1, -2,  0,  2,  1},
        { -4, -8,  0,  8,  4},
        { -6,-12,  0, 12,  6},
        { -4, -8,  0,  8,  4},
        { -1, -2,  0,  2,  1}}

    };

    const int tidx_z = threadIdx.x;
    const int tidx_x = threadIdx.y;
    const int tidx_y = threadIdx.z;
    const int bidx_z = blockIdx.x;
    const int bidx_x = blockIdx.y;
    const int bidx_y = blockIdx.z;
    const int bdim_z = blockDim.x;
    const int bdim_x = blockDim.y;
    const int bdim_y = blockDim.z;

    const int basez = bidx_z * bdim_z;
    const int basex = bidx_x * bdim_x;
    const int basey = bidx_y * bdim_y;
    const int z = basez + tidx_z;
    const int x = basex + tidx_x;
    const int y = basey + tidx_y;

    // if(bidx_x == 5 && bidx_y ==5 && bidx_z == 0){
    //     printf("%d\n", channels * ((width + 4) * y + x) + z);
    // }

    __shared__ unsigned char smSrc[3 * (BLOCK_N_X + 4) * (BLOCK_N_Y + 4)];


    if(x > width + 4 - 1 || y > height + 4 - 1) return;
    
    
    smSrc[channels * ((BLOCK_N_X + 4) * tidx_y + tidx_x) + tidx_z] =\
                        s[channels * ((width + 4) * y + x) + z];


    if((tidx_x < 4) && (BLOCK_N_X + x <= width + 4 - 1)){
        smSrc[channels * ((BLOCK_N_X + 4) * tidx_y + BLOCK_N_X + tidx_x) + tidx_z] =\
                        s[channels * ((width + 4) * y + BLOCK_N_X + x) + z];
    }

    if((tidx_y < 4) && (BLOCK_N_Y + y <= height + 4 - 1)){
        smSrc[channels * ((BLOCK_N_X + 4) * (BLOCK_N_Y + tidx_y) + tidx_x) + tidx_z] =\
                        s[channels * ((width + 4) * (BLOCK_N_Y + y) + x) + z];
    }

    if((tidx_x < 4) && (tidx_y < 4) &&\
                    (BLOCK_N_X + x <= width + 4 - 1) && (BLOCK_N_Y + y <= height + 4 - 1)){
        smSrc[channels * ((BLOCK_N_X + 4) * (BLOCK_N_Y + tidx_y) + BLOCK_N_X + tidx_x) + tidx_z] =\
                        s[channels * ((width + 4) * (BLOCK_N_Y + y) + BLOCK_N_X + x) + z];
    }
    


    if(x >= width || y >= height)return;

    __syncthreads();


    float val[2] = {0.0};

    for (int i = 0; i < MASK_N; ++i) {

        for (int v = 0; v <= 4; ++v) {     
            for (int u = 0; u <= 4; ++u) { 
                val[i] += smSrc[channels * ((BLOCK_N_X + 4) * (tidx_y + v)\
                                            + (tidx_x + u)) + tidx_z] * mask[i][u][v];

                // val[i] += s[channels * ((width + 4) * (y + v) + x + u) + z] * mask[i][u][v];
            }
        }
    }

    val[0] = sqrt(val[0]*val[0] + val[1]*val[1]) / SCALE;

    const unsigned char c = (val[0] > 255.0) ? 255 : val[0];

    t[channels * (width * y + x) + z] = c;
}









// __global__ void sobel(unsigned char *s, unsigned char *t, 
//                                 unsigned height, unsigned width, unsigned channels)
// {


//     int mask[MASK_N][MASK_X][MASK_Y] = {
    
//         {{ -1, -4, -6, -4, -1},
//         { -2, -8,-12, -8, -2},
//         {  0,  0,  0,  0,  0},
//         {  2,  8, 12,  8,  2},
//         {  1,  4,  6,  4,  1}},

//         {{ -1, -2,  0,  2,  1},
//         { -4, -8,  0,  8,  4},
//         { -6,-12,  0, 12,  6},
//         { -4, -8,  0,  8,  4},
//         { -1, -2,  0,  2,  1}}

//     };

//     int basex = blockIdx.x * blockDim.x;
//     int basey = blockIdx.y * blockDim.y;
//     int basez = blockIdx.z * blockDim.z;
    
//     int nextBasex = basex + blockDim.x;
//     int nextBasey = basey + blockDim.y;

//     int x = basex + threadIdx.x;
//     int y = basey + threadIdx.y;
//     int z = basez + threadIdx.z;


//     // if(blockIdx.x ==5 && blockIdx.y ==5 && blockIdx.z ==0){
//     //     printf("%d, %d\n", threadIdx.x, threadIdx.y);
//     // }


//     __shared__ unsigned char smSrc[(BLOCK_N_X + 4) * (BLOCK_N_Y + 4)];


//     if(x > width + 4 - 1 || y > height + 4 - 1) return;
    
//     smSrc[(BLOCK_N_X + 4) * threadIdx.y + threadIdx.x] = s[channels * ((width + 4) * y + x) + z];


//     if((threadIdx.x < 4) && (BLOCK_N_X + x <= width + 4 - 1)){
//         smSrc[(BLOCK_N_X + 4) * threadIdx.y + BLOCK_N_X + threadIdx.x] =\
//                         s[channels * ((width + 4) * y + BLOCK_N_X + x) + z];
//     }

//     if((threadIdx.y < 4) && (BLOCK_N_Y + y <= height + 4 - 1)){
//         smSrc[(BLOCK_N_X + 4) * (BLOCK_N_Y + threadIdx.y) + threadIdx.x] =\
//                         s[channels * ((width + 4) * (BLOCK_N_Y + y) + x) + z];
//     }

//     if((threadIdx.x < 4) && (threadIdx.y < 4) &&\
//                      (BLOCK_N_X + x <= width + 4 - 1) && (BLOCK_N_Y + y <= height + 4 - 1)){
//         smSrc[(BLOCK_N_X + 4) * (BLOCK_N_Y + threadIdx.y) + BLOCK_N_X + threadIdx.x] =\
//                         s[channels * ((width + 4) * (BLOCK_N_Y + y) + BLOCK_N_X + x) + z];
//     }




//     if(x >= width || y >= height)return;

//     __syncthreads();


//     float val[2] = {0.0};

//     for (int i = 0; i < MASK_N; ++i) {

//         for (int v = 0; v <= 4; ++v) {     
//             for (int u = 0; u <= 4; ++u) { 
//                 val[i] += smSrc[(BLOCK_N_X + 4) * (threadIdx.y + v) + (threadIdx.x + u)]\
//                              * mask[i][u][v];

//                 // val[i] += s[channels * ((width + 4) * (y + v) + x + u) + z] * mask[i][u][v];
//             }
//         }
//     }

//     val[0] = sqrt(val[0]*val[0] + val[1]*val[1]) / SCALE;

//     const unsigned char c = (val[0] > 255.0) ? 255 : val[0];

//     t[channels * (width * y + x) + z] = c;

// }





int main(int argc, char** argv) {
    assert(argc == 3);
    
    
    unsigned height, width, channels, gridNx, gridNy;
    unsigned char *src_img = NULL;
    read_png(argv[1], &src_img, &height, &width, &channels);
    assert(channels == 3);
    printf("width x height: %d x %d\n", width, height);


    gridNx = width / BLOCK_N_X + 1;
    gridNy = height / BLOCK_N_Y + 1;
    dim3 nThreadsPerBlock(BLOCK_N_Z, BLOCK_N_X, BLOCK_N_Y);
    dim3 nBlocks(GRID_N_Z, gridNx, gridNy);

    unsigned char *devSrc, *devDst;
    hipMallocManaged(&devSrc, (height + 4) * (width + 4) * channels * sizeof(unsigned char));

    auto start = high_resolution_clock::now();

    hipMemcpy(devSrc, src_img, (height + 4) * (width + 4) * channels * sizeof(unsigned char), hipMemcpyHostToDevice);

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout<<"hipMemcpy src_img dt: "<<duration.count()<<" us"<<endl;

    
    hipMalloc(&devDst, height * width * channels * sizeof(unsigned char));


    start = high_resolution_clock::now();

    sobel<<<nBlocks, nThreadsPerBlock>>>(devSrc, devDst, height, width, channels); 

    hipDeviceSynchronize();

    stop = high_resolution_clock::now();
    duration = duration_cast<microseconds>(stop - start);
    cout<<"kernel dt: "<<duration.count()<<" us"<<endl;



    unsigned char* dst_img =
        (unsigned char*) malloc(height * width * channels * sizeof(unsigned char));

    hipMemcpy(dst_img, devDst, height * width * channels * sizeof(unsigned char), hipMemcpyDeviceToHost);

    write_png(argv[2], dst_img, height, width, channels);

    hipFree(devSrc);
    hipFree(devDst);
    free(src_img);
    free(dst_img);

    return 0;
}



