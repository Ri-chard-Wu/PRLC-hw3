
#include <hip/hip_runtime.h>
#include <png.h>
#include <zlib.h>

#include <cassert>
#include <cmath>
#include <cstdlib>
#include <iostream>

#define MASK_N 2
#define MASK_X 5
#define MASK_Y 5
#define SCALE 8

// clang-format off

int mask[MASK_N][MASK_X][MASK_Y] = {
  
    {{ -1, -4, -6, -4, -1},
     { -2, -8,-12, -8, -2},
     {  0,  0,  0,  0,  0},
     {  2,  8, 12,  8,  2},
     {  1,  4,  6,  4,  1}},

    {{ -1, -2,  0,  2,  1},
     { -4, -8,  0,  8,  4},
     { -6,-12,  0, 12,  6},
     { -4, -8,  0,  8,  4},
     { -1, -2,  0,  2,  1}}

};



// clang-format on

int read_png(const char* filename, unsigned char** image, unsigned* height, unsigned* width,
    unsigned* channels) {
    unsigned char sig[8];
    FILE* infile;
    infile = fopen(filename, "rb");

    fread(sig, 1, 8, infile);
    if (!png_check_sig(sig, 8)) return 1; /* bad signature */

    png_structp png_ptr;
    png_infop info_ptr;

    png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png_ptr) return 4; /* out of memory */

    info_ptr = png_create_info_struct(png_ptr);
    if (!info_ptr) {
        png_destroy_read_struct(&png_ptr, NULL, NULL);
        return 4; /* out of memory */
    }

    png_init_io(png_ptr, infile);
    png_set_sig_bytes(png_ptr, 8);
    png_read_info(png_ptr, info_ptr);
    int bit_depth, color_type;
    png_get_IHDR(png_ptr, info_ptr, width, height, &bit_depth, &color_type, NULL, NULL, NULL);

    png_uint_32 i, rowbytes;
    png_bytep row_pointers[*height];
    png_read_update_info(png_ptr, info_ptr);
    rowbytes = png_get_rowbytes(png_ptr, info_ptr);
    *channels = (int)png_get_channels(png_ptr, info_ptr);

    if ((*image = (unsigned char*)malloc(rowbytes * *height)) == NULL) {
        png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
        return 3;
    }

    for (i = 0; i < *height; ++i) {
        row_pointers[i] = *image + i * rowbytes;
    }

    png_read_image(png_ptr, row_pointers);
    png_read_end(png_ptr, NULL);
    png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
    return 0;
}



void write_png(const char* filename, png_bytep image, const unsigned height, const unsigned width,
    const unsigned channels) {
    FILE* fp = fopen(filename, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_set_IHDR(png_ptr, info_ptr, width, height, 8, PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
        PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);
    png_set_filter(png_ptr, 0, PNG_NO_FILTERS);
    png_write_info(png_ptr, info_ptr);
    png_set_compression_level(png_ptr, 0);

    png_bytep row_ptr[height];
    for (int i = 0; i < height; ++i) {
        row_ptr[i] = image + i * width * channels * sizeof(unsigned char);
    }
    png_write_image(png_ptr, row_ptr);
    png_write_end(png_ptr, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    fclose(fp);
}



void sobel(unsigned char* s, unsigned char* t, unsigned height, unsigned width, unsigned channels) {
  
    int x, y, i, v, u;
    int R, G, B;
    double val[MASK_N * 3] = {0.0};
    
    int adjustX, adjustY, xBound, yBound;
    adjustX = (MASK_X % 2) ? 1 : 0;
    adjustY = (MASK_Y % 2) ? 1 : 0;
    xBound = MASK_X / 2;
    yBound = MASK_Y / 2;

    for (y = 0; y < height; ++y) {

        for (x = 0; x < width; ++x) {

            for (i = 0; i < MASK_N; ++i) {

                val[i * 3 + 2] = 0.0;
                val[i * 3 + 1] = 0.0;
                val[i * 3] = 0.0;

                for (v = -yBound; v < yBound + adjustY; ++v) {
                    for (u = -xBound; u < xBound + adjustX; ++u) {
                        if ((x + u) >= 0 && (x + u) < width && y + v >= 0 && y + v < height) {
                            R = s[channels * (width * (y + v) + (x + u)) + 2];
                            G = s[channels * (width * (y + v) + (x + u)) + 1];
                            B = s[channels * (width * (y + v) + (x + u)) + 0];
                            val[i * 3 + 2] += R * mask[i][u + xBound][v + yBound];
                            val[i * 3 + 1] += G * mask[i][u + xBound][v + yBound];
                            val[i * 3 + 0] += B * mask[i][u + xBound][v + yBound];
                        }
                    }
                }
            }

            double totalR = 0.0;
            double totalG = 0.0;
            double totalB = 0.0;
            for (i = 0; i < MASK_N; ++i) {
                totalR += val[i * 3 + 2] * val[i * 3 + 2];
                totalG += val[i * 3 + 1] * val[i * 3 + 1];
                totalB += val[i * 3 + 0] * val[i * 3 + 0];
            }

            totalR = sqrt(totalR) / SCALE;
            totalG = sqrt(totalG) / SCALE;
            totalB = sqrt(totalB) / SCALE;
            const unsigned char cR = (totalR > 255.0) ? 255 : totalR;
            const unsigned char cG = (totalG > 255.0) ? 255 : totalG;
            const unsigned char cB = (totalB > 255.0) ? 255 : totalB;
            t[channels * (width * y + x) + 2] = cR;
            t[channels * (width * y + x) + 1] = cG;
            t[channels * (width * y + x) + 0] = cB;
        }
        
    }
    
}

// __global__ void sobel()
// {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     int j = blockIdx.y * blockDim.y + threadIdx.y;
//     C[j][i] = A[j][i] * doubleValue(B[j][i]);
// }



// 1.png: 4928 x 3264
// 2.png: 16320 x 10809
// 3.png: 634 x 634
// 4.png: 900 x 622
// 5.png: 1800 x 1244
// 6.png: 3600 x 2488
// 7.png: 7200 x 4976
// 8.png: 14400 x 9952



int main(int argc, char** argv) {
    assert(argc == 3);

    unsigned height, width, channels;
    unsigned char* src_img = NULL;

    read_png(argv[1], &src_img, &height, &width, &channels);
    assert(channels == 3);

    printf("width x height: %d x %d\n", width, height);

    unsigned char* dst_img =
        (unsigned char*)malloc(height * width * channels * sizeof(unsigned char));

    sobel(src_img, dst_img, height, width, channels);

    write_png(argv[2], dst_img, height, width, channels);

    free(src_img);
    free(dst_img);

    return 0;
}
