
#include <hip/hip_runtime.h>
#include <png.h>
#include <zlib.h>

#include <cassert>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <chrono>

using namespace std::chrono;
using namespace std;

#define MASK_N 2
#define MASK_X 5
#define MASK_Y 5
#define SCALE 8



#define CC_GRID_MAX_X_DIM (1 << 31) - 1
#define CC_GRID_MAX_Y_DIM 65535
#define CC_GRID_MAX_Z_DIM 65535

#define CC_BLOCK_MAX_X_DIM 1024
#define CC_BLOCK_MAX_Y_DIM 1024
#define CC_BLOCK_MAX_Z_DIM 64

#define CC_BLOCK_MAX_N_THREADS 1024

#define CC_MAX_N_RSD_BLOCKS 32
#define CC_MAX_N_RSD_WARPS 64
#define CC_MAX_N_RSD_THREADS 2048


// 1.png: 4928 x 3264 x 3
// 2.png: 16320 x 10809 x 3
// 3.png: 634 x 634 x 3

// 4.png: 900 x 622 x 3

// 5.png: 1800 x 1244 x 3
// 6.png: 3600 x 2488 x 3
// 7.png: 7200 x 4976 x 3
// 8.png: 14400 x 9952 x 3



#define GRID_N_Z 1

#define BLOCK_N_X 32
#define BLOCK_N_Y 4
#define BLOCK_N_Z 3






int read_png(const char* filename, unsigned char** image, unsigned* height, unsigned* width,
    unsigned* channels) {
    unsigned char sig[8];
    FILE* infile;
    infile = fopen(filename, "rb");

    fread(sig, 1, 8, infile);
    if (!png_check_sig(sig, 8)) return 1; /* bad signature */

    png_structp png_ptr;
    png_infop info_ptr;

    png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png_ptr) return 4; /* out of memory */

    info_ptr = png_create_info_struct(png_ptr);
    if (!info_ptr) {
        png_destroy_read_struct(&png_ptr, NULL, NULL);
        return 4; /* out of memory */
    }

    png_init_io(png_ptr, infile);
    png_set_sig_bytes(png_ptr, 8);
    png_read_info(png_ptr, info_ptr);
    int bit_depth, color_type;
    png_get_IHDR(png_ptr, info_ptr, width, height, &bit_depth, &color_type, NULL, NULL, NULL);

    png_uint_32 i, rowbytes;
    png_bytep row_pointers[*height];
    png_read_update_info(png_ptr, info_ptr);
    rowbytes = png_get_rowbytes(png_ptr, info_ptr);
    *channels = (int)png_get_channels(png_ptr, info_ptr);

    if ((*image = (unsigned char*)calloc((rowbytes + 4*3) * (*height + 4), 1)) == NULL) {
        png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
        return 3;
    }
 
    for (i = 0; i < *height; ++i) {
        row_pointers[i] = *image + (i + 2) * (rowbytes + 4*3) + 2*3;
    }

    png_read_image(png_ptr, row_pointers);
    png_read_end(png_ptr, NULL);
    png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
    return 0;
}


void write_png(const char* filename, png_bytep image, const unsigned height, const unsigned width,
    const unsigned channels) {
    FILE* fp = fopen(filename, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_set_IHDR(png_ptr, info_ptr, width, height, 8, PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
        PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);
    png_set_filter(png_ptr, 0, PNG_NO_FILTERS);
    png_write_info(png_ptr, info_ptr);
    png_set_compression_level(png_ptr, 0);

    png_bytep row_ptr[height];
    for (int i = 0; i < height; ++i) {
        row_ptr[i] = image + i * width * channels * sizeof(unsigned char);
    }
    png_write_image(png_ptr, row_ptr);
    png_write_end(png_ptr, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    fclose(fp);
}






__global__ void sobel(unsigned char *s, unsigned char *t, 
                                unsigned height, unsigned width, unsigned channels)
{

    short mask[MASK_N * MASK_X * MASK_Y] = {

          -1, -4, -6, -4, -1,
          -2, -8,-12, -8, -2,
           0,  0,  0,  0,  0,
           2,  8, 12,  8,  2,
           1,  4,  6,  4,  1,
         
          -1, -2,  0,  2,  1,
          -4, -8,  0,  8,  4,
          -6,-12,  0, 12,  6,
          -4, -8,  0,  8,  4,
          -1, -2,  0,  2,  1} ;

    const unsigned short tidx_z = threadIdx.x;
    const unsigned short tidx_x = threadIdx.y;
    const unsigned short tidx_y = threadIdx.z;
    const int bidx_z = blockIdx.x;
    const int bidx_x = blockIdx.y;
    const int bidx_y = blockIdx.z;
    const int bdim_z = blockDim.x;
    const int bdim_x = blockDim.y;
    const int bdim_y = blockDim.z;
    const int basez = bidx_z * bdim_z;
    const int basex = bidx_x * bdim_x;
    const int basey = bidx_y * bdim_y;
    const int z = basez + tidx_z;
    const int x = basex + tidx_x;
    const int y = basey + tidx_y; 

    int idx_raw, idx_divRound, y_group, y_id, char_batch_idx;

    unsigned long long idx_raw_64, idx_divRound_64;

    __shared__ unsigned char smSrc[128 * (BLOCK_N_Y + 4)];
    __shared__ unsigned short xzBase[BLOCK_N_Y + 4];

    // if(x > width + 4 - 1 || y > height + 4 - 1) return;
    


    if(tidx_x < 22 && tidx_z < 3 && tidx_y == 0 && 3 * tidx_x + tidx_z <= 63 ){ 
        
        y_group = (3 * tidx_x + tidx_z) / 32; 
        y_id =  ((3 * tidx_x + tidx_z) - y_group * 32 ) / 8;
        char_batch_idx = (3 * tidx_x + tidx_z) - y_group * 32 - y_id * 8;


        if((y_group * BLOCK_N_Y + basey + y_id) <= (height + 4 - 1)){
            idx_raw_64 = 3 * ((unsigned long long)(width + 4)) * ((unsigned long long)(y_group * BLOCK_N_Y + basey + y_id));
            idx_raw_64 += 3 * (unsigned long long)basex;
            idx_raw_64 += (unsigned long long)basez;

            idx_divRound_64 =  (idx_raw_64 >> 4);

            xzBase[y_group * BLOCK_N_Y + y_id] = idx_raw_64 - idx_divRound_64 * 16;
            idx_divRound_64 += char_batch_idx;

            reinterpret_cast<int4 *>(smSrc)[8 * (y_group * BLOCK_N_Y + y_id) +\
                                    char_batch_idx] = reinterpret_cast<int4 *>(s)[idx_divRound_64];
        }                                
    }


    if(x >= width || y >= height)return;

    __syncthreads();
   
    short val0 = 0;
    short val1 = 0;
    float result;
    unsigned char a;


    for (char u = 0; u <= 1; ++u) { 
        for (char v = 0; v <= 4; ++v) { 
            idx_raw = 128 * (tidx_y + v) + xzBase[tidx_y + v] + 3 * (tidx_x + u) + tidx_z;
            a = smSrc[idx_raw];
            val0 += a * mask[5 * u + v];
        }
    }

    for (char u = 3; u <= 4; ++u) { 
        for (char v = 0; v <= 4; ++v) { 
            idx_raw = 128 * (tidx_y + v) + xzBase[tidx_y + v] + 3 * (tidx_x + u) + tidx_z;
            a = smSrc[idx_raw];
            val0 += a * mask[5 * u + v];
        }
    }

    
    for (char u = 0; u <= 4; ++u) { 
        for (char v = 0; v <= 1; ++v) {     
            idx_raw = 128 * (tidx_y + v) + xzBase[tidx_y + v] + 3 * (tidx_x + u) + tidx_z;
            a = smSrc[idx_raw];
            val1 += a * mask[25 + 5 * u + v];
        }

        for (char v = 3; v <= 4; ++v) {     
            idx_raw = 128 * (tidx_y + v) + xzBase[tidx_y + v] + 3 * (tidx_x + u) + tidx_z;
            a = smSrc[idx_raw];
            val1 += a * mask[25 + 5 * u + v];
        }
    }


    result = sqrtf(((int)val0) * ((int)val0) + ((int)val1)*((int)val1)) / SCALE;
    const unsigned char c = (result > 255.0) ? 255 : result;

    t[3 * (width * y + x) + z] = c;

}


int main(int argc, char** argv) {
    assert(argc == 3);
    
    
    unsigned height, width, channels, gridNx, gridNy;
    unsigned char *src_img = NULL;
    unsigned char *devSrc, *devDst;
    

    auto start = high_resolution_clock::now();
    read_png(argv[1], &src_img, &height, &width, &channels);
    hipMalloc(&devSrc, (height + 4) * (width + 4) * channels * sizeof(unsigned char));
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout<<"read_png() time: "<<duration.count()<<" us"<<endl;


    assert(channels == 3);
    // printf("width x height: %d x %d\n", width, height);


    gridNx = width / BLOCK_N_X + 1;
    gridNy = height / BLOCK_N_Y + 1;
    dim3 nThreadsPerBlock(BLOCK_N_Z, BLOCK_N_X, BLOCK_N_Y);
    dim3 nBlocks(GRID_N_Z, gridNx, gridNy);

    
    

    start = high_resolution_clock::now();
    hipMemcpy(devSrc, src_img, (height + 4) * (width + 4) * channels * sizeof(unsigned char), hipMemcpyHostToDevice);
    stop = high_resolution_clock::now();
    duration = duration_cast<microseconds>(stop - start);
    cout<<"hipMemcpyHostToDevice time: "<<duration.count()<<" us"<<endl;


    hipMalloc(&devDst, height * width * channels * sizeof(unsigned char));


    start = high_resolution_clock::now();
    sobel<<<nBlocks, nThreadsPerBlock>>>(devSrc, devDst, height, width, channels); 
    hipDeviceSynchronize();
    stop = high_resolution_clock::now();
    duration = duration_cast<microseconds>(stop - start);
    cout<<"kernel time: "<<duration.count()<<" us"<<endl;

    unsigned char* dst_img =
        (unsigned char*) malloc(height * width * channels * sizeof(unsigned char));
                

    start = high_resolution_clock::now();
    hipMemcpy(dst_img, devDst, height * width * channels * sizeof(unsigned char), hipMemcpyDeviceToHost);
    stop = high_resolution_clock::now();
    duration = duration_cast<microseconds>(stop - start);
    cout<<"hipMemcpyDeviceToHost time: "<<duration.count()<<" us"<<endl;



    start = high_resolution_clock::now();
    write_png(argv[2], dst_img, height, width, channels);
    stop = high_resolution_clock::now();
    duration = duration_cast<microseconds>(stop - start);
    cout<<"write_png() time: "<<duration.count()<<" us"<<endl;


    hipFree(devSrc);
    hipFree(devDst);
    // free(src_img);
    free(dst_img);

    return 0;
}


